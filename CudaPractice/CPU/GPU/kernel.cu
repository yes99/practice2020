#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"


__global__ void AddVecGPU(float* c, float* a, float* b)
{
	//for���� �������. ���� ������ ����ȭ �ƴ�
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
	printf("Tread Id = %d\n", i);
}


int main()
{
	int N = 1024 * 1024;
	float* a = new float[N];
	float* b = new float[N];
	float* c = new float[N];
	for (int i = 0; i < N; ++i)
	{
		//rand() : 0~ RAND MAX
		a[i] = rand() / (float)RAND_MAX; //0.0 ~ 1.0 ������ ������ ��
		b[i] = -a[i];
		c[i] = 0.0;
	}


	// 1. ����� gpu����
	hipError_t cudaStatus = hipSetDevice(0);

	// 2. gpu(Device) �� �޸𸮸� �������� �Ҵ��Ѵ�.
	float* dev_a, * dev_b, * dev_c;
	hipMalloc((void**)&dev_a, sizeof(float) * N);
	hipMalloc((void**)&dev_b, sizeof(float) * N);
	hipMalloc((void**)&dev_c, sizeof(float) * N);

	// 3. cpu �迭�� �����͸� gpu �迭�� �����Ų��
	hipMemcpy(dev_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, sizeof(float) * N, hipMemcpyHostToDevice);

	// 4. gpu ���� ����� �Լ�(= Ŀ��, kernel)�� ȣ��
	AddVecGPU<<<1, N>>>(dev_c, dev_a, dev_b);

	//��� �����尡 ������ ������ �� ���� ��ٸ���. 
	hipDeviceSynchronize();

	// 5. ����� GPU �޸𸮿��� CPU �޸𸮷� �����Ѵ�.
	hipMemcpy(c, dev_c, sizeof(float) * N, hipMemcpyDeviceToHost);



	
	for (int i = 0; i < N; ++i)
	{
//		printf("c[%d] = %f\n", i, c[i]);

		if (c[i] != 0.0)
			printf("Error\n");

	}

	delete[] a;
	delete[] b;
	delete[] c;
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);




}