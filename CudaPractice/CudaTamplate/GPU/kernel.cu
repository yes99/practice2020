#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "..\usr\include\GL\freeglut.h"
#include <stdio.h>
#include <time.h>
#include <math.h>

//�ݹ� �Լ�
void Render(); // �̹����� ũ��
void Reshape(int w, int h); // �迭 1024 by 1024  rgb �̹��� �̱� ����
void Timer(int id);

//����� ���� �Լ�
void CreateJuliaSet();

//Ŀ�� �Լ�
__global__ void julia_kernel(unsigned char* pImage, float cx, float cy, float R, int MaxIter);
__device__ int Julia(float a, float b, float cx, float cy, float R, int MaxIter);
__device__ void GetColorRainbow(float t, int& r, int& g, int& b);

#define TILE_WIDTH 32
const int Dim = 1024;
unsigned char Image[Dim*Dim * 3];
unsigned char *DevImage;
float theta = 0.0;

int main(int argc, char **argv)
{
	//GLUT �ʱ�ȭ
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB);

	//������ ũ�� ���� �� ����
	glutInitWindowSize(Dim, Dim);
	glutCreateWindow("Julia Set(GPU)");

	//�ݹ� �Լ� ���
	glutDisplayFunc(Render);
	glutReshapeFunc(Reshape);
	glutTimerFunc(1, Timer, 0);

	//GPU �޸� ���� �Ҵ�
	hipSetDevice(0);
	hipMalloc((void **)&DevImage, sizeof(unsigned char) * Dim * Dim * 3);

	//�̺�Ʈ ó�� ���� ����
	glutMainLoop();

	//�޸� ����
	hipFree(DevImage);
	hipDeviceReset();
	return 0;
}

void Render()
{
	//�ȼ� ����(���)�� ������� �ʱ�ȭ�Ѵ�.
	glClearColor(1, 1, 1, 1);
	glClear(GL_COLOR_BUFFER_BIT);

	//Julia ���� ã�� �ȼ� ���۸� ä���.
	CreateJuliaSet();
	glDrawPixels(Dim, Dim, GL_RGB, GL_UNSIGNED_BYTE, Image);
	glFinish();
}

void Reshape(int w, int h)
{
	glViewport(0, 0, w, h);
}

void Timer(int id)
{
	theta += 0.01;
	glutPostRedisplay();
	glutTimerFunc(1, Timer, 0);
}


void CreateJuliaSet()
{
	float cx = cos(theta) * 0.7885;
	float cy = sin(theta) * 0.7885;
	float R = sqrt(cx * cx + cy * cy) * 3;
	int MaxIter = 256; //��ȭ�� �ݺ�
	
	//�׸��� ũ��� ����� ũ�� ����
	dim3 gridDim(Dim / TILE_WIDTH, Dim / TILE_WIDTH);
	dim3 blockDim(TILE_WIDTH, TILE_WIDTH);

	// Ŀ���Լ� (julia_kernel) ȣ��
	clock_t st = clock();
	julia_kernel << <gridDim, blockDim >> > (DevImage, cx, cy, R, MaxIter);
	hipDeviceSynchronize();

	//����� CPU�� �����Ѵ�
	hipMemcpy(Image, DevImage, sizeof (unsigned char)* Dim* Dim * 3, hipMemcpyDeviceToHost);
	printf("Elapsed time = %u ms\n", clock() - st);
}

__global__ void julia_kernel(unsigned char* pImage, float cx, float cy, float R, int MaxIter)
{
	int x = blockIdx.x * TILE_WIDTH + threadIdx.x;
	int y = blockIdx.y * TILE_WIDTH + threadIdx.y;
	if (x < Dim && y < Dim) // �̹��� ũ�� �ȿ� ���ٸ�
	{
		float a = (x / (float)(Dim - 1) * 2.0 - 1.0) * R * 0.5;
		float b = -(y/ (float)(Dim - 1) * 2.0 - 1.0) * R * 0.5;

		int offset = (y * Dim + x) * 3;
		int Iter = Julia(a, b, cx, cy, R, MaxIter); // z(0)�� �ٸ��ƿ� ���ԵǴ��� Ȯ��
		
		float t = (float)Iter / MaxIter; //(0~MaxIter) -> (0,1)
		int R, G, B;
		GetColorRainbow(t, R, G, B);
		pImage[offset] = R;
		pImage[offset + 1] = G;
		pImage[offset + 2] = B;
	}
}


__device__ int Julia(float a, float b, float cx, float cy, float R, int MaxIter)
{
	for (int k = 0; k < MaxIter; ++k)
	{
		float re = a * a - b * b;
		float im = 2 * a * b;
		a = re + cx;
		b = im + cy;
		
		float len = sqrtf(a * a + b * b);
		if (a * a + b * b > R * R)
			return k;
	}
	return MaxIter;

}

__device__ void GetColorRainbow(float t, int& r, int& g, int& b)
{
	int X = (int)(6 * t);
	// 0.0 <= t <= 1.0
	t = 6.0 * t - X;
	switch (X)
	{
	case 0:	// ���� ~ ��Ȳ
		r = 255;
		g = t * 127;
		b = 0;
		//r = g = b = 255;
		break;

	case 1:	// ��Ȳ ~ ���
		r = 0;
		g = (1.0 - t) * 127 + t * 255;
		b = 0;
		break;

	case 2:	// ��� ~ �ʷ�
		r = (1.0 - t) * 255;
		g = 255;
		b = 0;
		break;

	case 3:	// �ʷ� ~ �Ķ�
		r = 0;
		g = (1.0 - t) * 255;
		b = t * 255;
		break;

	case 4:	// �Ķ� ~ ����
		r = t * 75;
		g = 0;
		b = (1.0 - t) * 255 + t * 130;
		break;

	case 5:	// ���� ~ ����
		r = (1.0 - t) * 75 + t * 148;
		g = 0;
		b = (1.0 - t) * 130 + t * 211;
		break;

	case 6:	// ����
		r = 148;
		g = 0;
		b = 211;
		break;
	}

	// ���� ���� �������� �Ҵ��Ѵ�.
	if (t < 0.0 || t > 1.0)
		r = g = b = 0;
}
