#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""


__global__ void MultMatGPU(float* P, float* M, float* N, int width)
{
	int i = threadIdx.y;
	int j = threadIdx.x;

	float sum = 0.0;
	for (int k = 0; k < width; ++k)
	{
		float a = M[i * width + k];
		float b = N[k * width + j];
		sum += a * b;
	}
}

void PrintMatrix(float* M, int width)
{
	for (int i = 0; i < width; ++i)
	{
		for (int j = 0; j < width; ++j)
		{
			printf("%3.0f   ", M[i * width + j]);
		}
		printf("\n");
	}
	printf("\n");
}

int main()
{
	//cpu �޸� ���� �Ҵ� �� �ʱ����
	int width = 32;
	float* P = new float[width * width];
	float* M = new float[width * width];
	float* N = new float[width * width];
	for (int i = 0; i < width * width; ++i)
	{
		// rand() : 0~ RAND_MAX
		M[i] = rand() % 3 - 1.0;
		N[i] = rand() % 3 - 1.0;
		P[i] = 0.0;
	}
	
	//����� ��gpu����
	hipSetDevice(0);

	//gpu�޸� �Ҵ�(1)
	float* dev_P, * dev_M, * dev_N;
	hipMalloc((void**)&dev_P, width * width * sizeof(float));
	hipMalloc((void**)&dev_M, width * width * sizeof(float));
	hipMalloc((void**)&dev_N, width * width * sizeof(float));

	//cpu���� gpu�� �޵����� ����(2)
	hipMemcpy(dev_M, M, width * width * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_N, N, width * width * sizeof(float), hipMemcpyHostToDevice);

	//Ŀ�� �Լ��� ȣ���Ͽ� ���������� ����� ���Ѵ�
	dim3 dimGrid(1, 1);
	dim3 dimBlock(width, width);
	clock_t st = clock();

	//������ ���� �� Ŀ�� �Լ� ȣ��
	MultMatGPU<<<dimGrid, dimBlock>>>MultMatGPU(dev_P, dev_M, dev_N, width);
	//�׸���� ����� �迭, �ϳ��� �׸��� �ȿ��� �������� ���.
	//��� �ȿ��� �������� �����尡 ����ִ�.

	//ù��° �Ķ���� : �׸��� �� ��ϼ�
	//�ι��� �Ķ���� : ��� ���� ������ ��
	hipDeviceSynchronize(); // ��� ��������� �����Ҷ� ���� ��ٸ�.
	printf("elapsed time = %u ms\n", clock() - st);

	hipMemcpy(P, dev_P, width * width * sizeof(float), hipMemcpyDeviceToHost);

	PrintMatrix(M, width);
	PrintMatrix(N, width);
	PrintMatrix(P, width);

	//�Ҵ�� gpu�޸� ����
	delete[] M;
	delete[] N;
	delete[] P;

	hipFree(dev_P);
	hipFree(dev_M);
	hipFree(dev_N);
}

























/*
__global__ void AddVecGPU(float* c, float* a, float* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];

}


int main()
{
	int N = 32;
	float* a = new float[N];
	float* b = new float[N];
	float* c = new float[N];
	for (int i = 0; i < N; ++i)
	{
		// rand() : 0~ RAND_MAX
		a[i] = rand() / (float)RAND_MAX; // 0.0 ~ 1.0 ������ ������ ��
		b[i] = -a[i];
		c[i] = 0.0;
	}

	// 1. ����� gpu�� �����Ѵ�
	hipSetDevice(0);
	hipError_t cudaStatus = hipSetDevice(0);

	// 2. GPU(Device)�� �޸𸮸� �������� �Ҵ��Ѵ�.
	float* dev_a, * dev_b, * dev_c;
	hipMalloc((void**)&dev_a, sizeof(float) * N);
	hipMalloc((void**)&dev_b, sizeof(float) * N);
	hipMalloc((void**)&dev_c, sizeof(float) * N);

	// 3. CPU �迭�� �����͸� GPU �迭�� �����Ѵ�.
	hipMemcpy(dev_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, sizeof(float) * N, hipMemcpyHostToDevice);

	// 4. GPU���� ����� �Լ��� ȣ��
	AddVecGPU<<<1, N>>>(dev_c, dev_a, dev_b);

	//��� �����尡 ������ ������ ������ ��ٸ���. 
	hipDeviceSynchronize();

	// 5. ����� GPU�޸𸮿��� CPU�޸𸮷� �����Ѵ�.
	hipMemcpy(c, dev_c, sizeof(float) * N, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; ++i)
	{
		printf("c[%d] = %f\n", i, c[i]);
		//if (c[i] != 0.0)
		//	printf("Error\n");

	}
	delete[] a;
	delete[] b;
	delete[] c;

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

}

*/